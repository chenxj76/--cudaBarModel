#include "hip/hip_runtime.h"
#include "bar.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void GPUInitOnDevice(double*u,double*v,int x,int y);
__global__ void GPUCalcBar(double*u,double*v,double*u0,double*v0,double*du,double*dv);

void Manage_Memory(int phase,double**u,double**v,double**u0,double**v0,double**dd_u,double**dd_v,double**dd_u0,double**dd_v0,double**dd_du,double**dd_dv){
	hipError_t Error;
	size_t size=nx*ny*sizeof(double);
	if (phase==1){
		*u=(double*)malloc(size);
		*v=(double*)malloc(size);
		*u0=(double*)malloc(size);
		*v0=(double*)malloc(size);
		Error=hipMalloc((void**)dd_u,size);
		Error=hipMalloc((void**)dd_u0,size);
		Error=hipMalloc((void**)dd_v,size);
		Error=hipMalloc((void**)dd_v0,size);
		Error=hipMalloc((void**)dd_du,size);
		Error=hipMalloc((void**)dd_dv,size);
		printf("MemoryMalloc:%s\n",hipGetErrorString(Error));
	}
	if (phase==2){
		free(*u);
		free(*v);
		free(*u0);
		free(*v0);
		Error=hipFree(*dd_u);
		Error=hipFree(*dd_u0);
		Error=hipFree(*dd_v);
		Error=hipFree(*dd_v0);
		Error=hipFree(*dd_du);
		Error=hipFree(*dd_dv);
	}

}

void Manage_Comms(int phase,double*hh_u,double*dd_u){

	hipError_t Error;
	size_t size=nx*ny*sizeof(double);
	if (phase==2){				
	hipMemcpy(hh_u,dd_u,size,hipMemcpyDeviceToHost);
		Error=hipMemcpy(hh_u,dd_u,size,hipMemcpyDeviceToHost);
		printf("device to host:%s\n",hipGetErrorString(Error));
		}
	if (phase==1){
		Error=hipMemcpy(dd_u,hh_u,size,hipMemcpyHostToDevice);
		printf("host to device:%s\n",hipGetErrorString(Error));}
}

__global__ void GPUInitOnDevice(double*u,double*v,int x,int y){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;//matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*x+ ix;//globalIdx
	//***init conditions*****			
		if(ix<x&&iy<y){
			u[idx] = 0.0; 
			v[idx] = 0.0;			
	}
}
void Call_GPU_Init(double*u0,double*v0,int x,int y){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUInitOnDevice<<<grid,block>>>(u0,v0,x,y);
	Error=hipDeviceSynchronize();
	printf("InitDeviceSynchronize:%s\n",hipGetErrorString(Error));
}

__global__ void GPUCalcBar(double*u,double*v,double*u0,double*v0,double*du,double*dv){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx
		double*hh_u0;
		double sldx, sldy; 
		int ncount;
		sldx = (1.0 / dx)/ dx;
		sldy = (1.0 / dy)/ dy;

	for (ncount = 1; ncount < 10001; ncount++){

		//****no flux boundary conditions*****
				u0[iy*nx] = u0[1+iy*nx];
				u0[nx+iy*nx] = u0[(nx-1)+iy*nx];
				v0[iy*nx] = v0[1+iy*nx];
				v0[nx+iy*nx] = v0[(nx-1)+iy*nx];
				u0[ix] = u0[ix+nx];
				u0[ix+ny*nx] = u0[ix+(ny-1)*nx];
				v0[ix] = v0[ix+nx];
				v0[ix+ny*nx] = v0[ix+(ny-1)*nx];		
				
		//*********** Center Differnce for Space *******
		
		
			du[idx] = -u0[idx]*(u0[idx] - 1.0)*(u0[idx] - (v0[idx] + bb) / aa) / eps + D*(sldx*(u0[(ix+1)+iy*nx] + u0[(ix-1)+iy*nx] - 2 * u0[ix+iy*nx]) 
									+ sldy*(u0[ix+(iy+1)*nx] + u0[ix+(iy-1)*nx] - 2 * u0[ix+iy*nx]));
			
			{
			if (u0[idx]< 1.0/3.0)
			dv[idx] = -v0[idx];
			else if(u0[idx]<1.0 || u0[idx]==1.0)
			dv[idx] = 1.0 - 6.75*u0[idx]*(u0[idx] - 1.0)*(u0[idx] - 1.0) - v0[idx];
			else
			dv[idx] = 1.0 - v0[idx];
			}

			
		//*****stimulation with a plane waves****
		if (ncount < 10 && ix<3 ){
				du[idx] = du[idx] + 1.5*cos(w*ncount*h) + 0.5;
		}

		//***********Forward Euler for Time ******
				u[idx] = u0[idx] + h*du[idx];
				v[idx] = v0[idx] + h*dv[idx];
		//***********Update the tow variabls******
				u0[idx] = u[idx];
				v0[idx] = v[idx];		
		
		//***********trancation 1/2 of the plane wave to generate a spiral wave******
		if (ncount == 1000 && idx<nx+1 && iy<(ny/2)+1){
				u0[idx] = 0;
				v0[idx] = 0;
		}
		if (ncount == 5000){
		 *hh_u0=*u0;
		}
	
	}	
}
void Call_GPU_Calc_Bar(double*u,double*v,double*u0,double*v0,double*du,double*dv){
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUCalcBar<<<grid,block>>>(u,v,u0,v0,du,dv);
	Error=hipDeviceSynchronize();
	printf("CalcDeviceSynchronize:%s\n",hipGetErrorString(Error));
}




