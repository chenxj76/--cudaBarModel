#include "hip/hip_runtime.h"
#include "bar.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
double*h_u,*h_v,*h_u0,*h_v0,*h_du,*h_dv;
double*d_u,*d_v,*d_u0,*d_v0,*d_du,*d_dv;
__global__ void GPUInitOnDevice(double*u,double*v,int x,int y);
__global__ void GPUCalcBar(double*u,double*v,double*u0,double*v0,double*du,double*dv);

void Manage_Memory(int phase,double**h_u,double**h_v,double**h_u0,double**h_v0,double**d_u,double**d_v,double**d_u0,double**d_v0,double**d_du,double**d_dv){
	hipError_t Error;
	size_t size=nx*ny*sizeof(double);
	if (phase==1){
		*h_u=(double*)malloc(size);
		*h_v=(double*)malloc(size);
		*h_u0=(double*)malloc(size);
		*h_v0=(double*)malloc(size);
		Error=hipMalloc((void**)d_u,size);
		Error=hipMalloc((void**)d_u0,size);
		Error=hipMalloc((void**)d_v,size);
		Error=hipMalloc((void**)d_v0,size);
		Error=hipMalloc((void**)d_du,size);
		Error=hipMalloc((void**)d_dv,size);
		printf("MemoryMalloc:%s\n",hipGetErrorString(Error));
	}
	if (phase==2){
		free(*h_u);
		free(*h_v);
		free(*h_u0);
		free(*h_v0);
		Error=hipFree(*d_u);
		Error=hipFree(*d_u0);
		Error=hipFree(*d_v);
		Error=hipFree(*d_v0);
		Error=hipFree(*d_du);
		Error=hipFree(*d_dv);
	}

}

void Manage_Comms(int phase,double**h_u0,double**d_u0){

	hipError_t Error;
	size_t size=nx*ny*sizeof(double);
	if (phase==2){				
		Error=hipMemcpy(*h_u0,*d_u0,size,hipMemcpyDeviceToHost);
		printf("device to host:%s\n",hipGetErrorString(Error));
		}
	if (phase==1){
		Error=hipMemcpy(*d_u0,*h_u0,size,hipMemcpyHostToDevice);
		printf("host to device:%s\n",hipGetErrorString(Error));}
}

__global__ void GPUInitOnDevice(double*u,double*v,int x,int y){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;//matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*x+ ix;//globalIdx
	//***init conditions*****			
		if(ix<x&&iy<y){
			u[idx] = 0.0; 
			v[idx] = 0.0;			
	}
}
void Call_GPU_Init(double**d_u0,double**d_v0,int x,int y){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUInitOnDevice<<<grid,block>>>(*d_u0,*d_v0,x,y);
	Error=hipDeviceSynchronize();
	printf("InitDeviceSynchronize:%s\n",hipGetErrorString(Error));
}

__global__ void GPUCalcBar(double*u,double*v,double*u0,double*v0,double*du,double*dv){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx

		double sldx, sldy; 
		int ncount;
		sldx = (1.0 / dx)/ dx;
		sldy = (1.0 / dy)/ dy;

	for (ncount = 1; ncount < 10001; ncount++){

		//****no flux boundary conditions*****
				u0[iy*nx] = u0[1+iy*nx];
				u0[nx+iy*nx] = u0[(nx-1)+iy*nx];
				v0[iy*nx] = v0[1+iy*nx];
				v0[nx+iy*nx] = v0[(nx-1)+iy*nx];
				u0[ix] = u0[ix+nx];
				u0[ix+ny*nx] = u0[ix+(ny-1)*nx];
				v0[ix] = v0[ix+nx];
				v0[ix+ny*nx] = v0[ix+(ny-1)*nx];		
				
		//*********** Center Differnce for Space *******
		
		
			du[idx] = -u0[idx]*(u0[idx] - 1.0)*(u0[idx] - (v0[idx] + bb) / aa) / eps + D*(sldx*(u0[(ix+1)+iy*nx] + u0[(ix-1)+iy*nx] - 2 * u0[ix+iy*nx]) 
									+ sldy*(u0[ix+(iy+1)*nx] + u0[ix+(iy-1)*nx] - 2 * u0[ix+iy*nx]));
			
			{
			if (u0[idx]< 1.0/3.0)
			dv[idx] = -v0[idx];
			else if(u0[idx]<1.0 || u0[idx]==1.0)
			dv[idx] = 1.0 - 6.75*u0[idx]*(u0[idx] - 1.0)*(u0[idx] - 1.0) - v0[idx];
			else
			dv[idx] = 1.0 - v0[idx];
			}

			
		//*****stimulation with a plane waves****
		if (ncount < 10 && ix<3 ){
				du[idx] = du[idx] + 1.5*cos(w*ncount*h) + 0.5;
		}

		//***********Forward Euler for Time ******
				u[idx] = u0[idx] + h*du[idx];
				v[idx] = v0[idx] + h*dv[idx];
		//***********Update the tow variabls******
				u0[idx] = u[idx];
				v0[idx] = v[idx];		
		
		//***********trancation 1/2 of the plane wave to generate a spiral wave******
		if (ncount == 1000 && idx<nx+1 && iy<(ny/2)+1){
				u0[idx] = 0;
				v0[idx] = 0;
		}
	//	if (ncount == 5000){
	//	 h_u0[idx]=u0[idx];   //问题出在这CalcDeviceSynchronize:an illegal memory access was encountered！
	//	}
	
	}	
}
void Call_GPU_Calc_Bar(double**d_u,double**d_v,double**d_u0,double**d_v0,double**d_du,double**d_dv){
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUCalcBar<<<grid,block>>>(*d_u,*d_v,*d_u0,*d_v0,*d_du,*d_dv);
	Error=hipDeviceSynchronize();
	printf("CalcDeviceSynchronize:%s\n",hipGetErrorString(Error));
}




