#include "hip/hip_runtime.h"
#include "bar.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
extern double*h_u,*h_v,*h_u0,*h_v0,*h_du,*h_dv;
extern double*d_u,*d_v,*d_u0,*d_v0,*d_du,*d_dv;


__global__ void GPUInitOnDevice(int x,int y,double*d_u,double*d_v);
__global__ void GPUBoundary(double*d_u,double*d_v);
__global__ void GPUCalcSpace(double*d_u0,double*d_v0,double*d_du,double*d_dv);
__global__ void GPUPlaneWave(double*d_du,int ncount);
__global__ void GPUForEuler(double*d_u,double*d_u0,double*d_du,double*d_v,double*d_v0,double*d_dv);
__global__ void GPUUpdate(double*d_u,double*d_u0,double*d_v,double*d_v0);
__global__ void GPUTrancation(double*d_u,double*d_v);


//----------------Init-------------------
__global__ void GPUInitOnDevice(double*d_u,double*d_v){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;//matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx

	//***init conditions*****			
		if(ix<nx+2&&iy<nx+2){//条件控制有多少个点赋初值
		unsigned int idx = iy*(nx+2)+ ix;//globalIdx	
			d_u[idx] = 0.0; 
			d_v[idx] = 0.0;			
	}
}
void Call_GPU_Init(){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+2+block.x-1)/block.x,(ny+2+block.y-1)/block.y);
	hipError_t Error;
	GPUInitOnDevice<<<grid,block>>>(d_u,d_v);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("InitDeviceSynchronize:%s\n",hipGetErrorString(Error));
}
//----------------boundary-------------------
__global__ void GPUBoundary(double*d_u,double*d_v){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
//unsigned int id = iy*(nx+2)+ ix;//globalIdx，包括4条边
		
		//****no flux boundary conditions*****
		if(ix<nx&&iy<nx){
				d_u[(iy+1)*(nx+2)+0] = d_u[(iy+1)*(nx+2)+1];
				d_u[(iy+1)*(nx+2)+(nx+1)] = d_u[(iy+1)*(nx+2)+nx];
				d_u[(ix+1)+0*(nx+2)] = d_u[(ix+1)+1*(nx+2)];
				d_u[(ix+1)+(ny+1)*(nx+2)] = d_u[(ix+1)+ny*(nx+2)];
				
				d_v[(iy+1)*(nx+2)+0] = d_v[(iy+1)*(nx+2)+1];
				d_v[(iy+1)*(nx+2)+(nx+1)] = d_v[(iy+1)*(nx+2)+nx];			
				d_v[(ix+1)+0*(nx+2)] = d_v[(ix+1)+1*(nx+2)];
				d_v[(ix+1)+(ny+1)*(nx+2)] = d_v[(ix+1)+ny*(nx+2)];
		}									
}		
void Call_GPU_Boun(){	
	dim3 block(BLOCK_SIZE,1);
	dim3 grid((nx+block.x-1)/block.x,1);
	hipError_t Error;
	GPUBoundary<<<grid,block>>>(d_u,d_v);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("GPUBoundarySynchronize:%s\n",hipGetErrorString(Error));
}			
			
//----------------Center Differnce for Space-------------------			
__global__ void GPUCalcSpace(double*d_u0,double*d_v0,double*d_du,double*d_dv){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
//unsigned int id=iy*(nx+2)+ix;//扩充了的global index
		double sldx, sldy; 
		sldx = (1.0 / dx)/ dx;
		sldy = (1.0 / dy)/ dy;

		if(ix<nx&&iy<ny){
			
			int idx = (iy+1)*(nx+2)+ (ix +1);//globalIdx,不要计算4条边界
			d_du[idx] = -d_u0[idx]*(d_u0[idx] - 1.0)*(d_u0[idx] - (d_v0[idx] + bb) / aa) / eps + D*(sldx*(d_u0[idx+1] + d_u0[idx-1] - 2 * d_u0[idx]) 
									+ sldy*(d_u0[idx+nx+2] + d_u0[idx-nx-2] - 2 * d_u0[idx]));
			
			
			if(d_u0[idx]< 1.0/3.0) d_dv[idx] = -d_v0[idx];
			else if(d_u0[idx]<1.0 || d_u0[idx]==1.0)
			d_dv[idx] = 1.0 - 6.75*d_u0[idx]*(d_u0[idx] - 1.0)*(d_u0[idx] - 1.0) - d_v0[idx];
			else d_dv[idx] = 1.0 - d_v0[idx];
						
				
		}
}		
void Call_GPU_Space(){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUCalcSpace<<<grid,block>>>(d_u0,d_v0,d_du,d_dv);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("GPUCalcSpaceSynchronize:%s\n",hipGetErrorString(Error));
}			

//----------------stimulation with a plane waves-------------------		
__global__ void GPUPlaneWave(double*d_du,int ncount){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx

	
		if (iy < nx && ix<3 ){
				
				int idx = (iy+1)*(nx+2)+ (ix +1);//globalIdx,不要计算4条边界
				d_du[idx] = d_du[idx] + 1.5*cos(w*ncount*h) + 0.5;
				
		}
}
void Call_GPU_PlaneWave(int ncount){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUPlaneWave<<<grid,block>>>(d_du,ncount);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("GPUPlaneWaveSynchronize:%s\n",hipGetErrorString(Error));
}
//----------------Forward Euler for Time-------------------
__global__ void GPUForEuler(double*d_u,double*d_u0,double*d_du,double*d_v,double*d_v0,double*d_dv){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx

		if(ix<nx&&iy<ny){
		
				int idx = (iy+1)*(nx+2)+ (ix +1);//globalIdx,不要计算4条边界	
				d_u[idx] = d_u0[idx] + h*d_du[idx];
				d_v[idx] = d_v0[idx] + h*d_dv[idx];
				}
}	
void Call_GPU_ForEuler(){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUForEuler<<<grid,block>>>(d_u,d_u0,d_du,d_v,d_v0,d_dv);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("GPUForEulerSynchronize:%s\n",hipGetErrorString(Error));
}			
//----------------Update the tow variabls-------------------
__global__ void GPUUpdate(double*d_u,double*d_u0,double*d_v,double*d_v0){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx

		if(ix<nx&&iy<ny){
				
				int idx = (iy+1)*(nx+2)+ (ix +1);//globalIdx,不要计算4条边界
				d_u0[idx] = d_u[idx];
				d_v0[idx] = d_v[idx];
				}
}
void Call_GPU_Update(){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUUpdate<<<grid,block>>>(d_u,d_u0,d_v,d_v0);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("GPUUpdateSynchronize:%s\n",hipGetErrorString(Error));
}				
//----------------trancation 1/2 of the plane wave to generate a spiral wave-------------------
__global__ void GPUTrancation(double*d_u0,double*d_v0){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx;//globalIdx
		if(ix<nx&&iy<(ny/2)+1){
				idx = (iy+1)*(nx+2)+ (ix +1);//globalIdx,不要计算4条边界			
				d_u0[idx] = 0.0;
				d_v0[idx] = 0.0;
		}
}
void Call_GPU_Trancation(){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUTrancation<<<grid,block>>>(d_u0,d_v0);
	Error=hipDeviceSynchronize();
	if (Error != hipSuccess)printf("GPUTrancationSynchronize:%s\n",hipGetErrorString(Error));
}








