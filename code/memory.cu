#include "bar.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

double*h_u,*h_v,*h_u0,*h_v0,*h_du,*h_dv;
double*d_u,*d_v,*d_u0,*d_v0,*d_du,*d_dv;


void Manage_Memory(int phase){
	hipError_t Error;
	size_t size=(nx+2)*(ny+2)*sizeof(double);
	if (phase==1){
		h_u=(double*)malloc(size);
		h_v=(double*)malloc(size);
		h_u0=(double*)malloc(size);
		h_v0=(double*)malloc(size);
		Error=hipMalloc((void**)&d_u,size);
		Error=hipMalloc((void**)&d_u0,size);
		Error=hipMalloc((void**)&d_v,size);
		Error=hipMalloc((void**)&d_v0,size);
		Error=hipMalloc((void**)&d_du,size);
		Error=hipMalloc((void**)&d_dv,size);
		printf("MemoryMalloc:%s\n",hipGetErrorString(Error));
	}
	if (phase==2){
		free(h_u);
		free(h_v);
		free(h_u0);
		free(h_v0);
		Error=hipFree(d_u);
		Error=hipFree(d_u0);
		Error=hipFree(d_v);
		Error=hipFree(d_v0);
		Error=hipFree(d_du);
		Error=hipFree(d_dv);
	}

}

void Manage_Comms(int phase){

	hipError_t Error;
	size_t size=(nx+2)*(ny+2)*sizeof(double);
	if (phase==2){				
		Error=hipMemcpy(h_u0,d_u0,size,hipMemcpyDeviceToHost);
		if (Error != hipSuccess)printf("device to host:%s\n",hipGetErrorString(Error));
		}
	if (phase==1){
		Error=hipMemcpy(d_u0,h_u0,size,hipMemcpyHostToDevice);
		if (Error != hipSuccess)printf("host to device:%s\n",hipGetErrorString(Error));}
}
