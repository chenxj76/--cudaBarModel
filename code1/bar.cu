#include "hip/hip_runtime.h"
#include "bar.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
double*h_u,*h_v,*h_u0,*h_v0,*h_du,*h_dv;
double*d_u,*d_v,*d_u0,*d_v0,*d_du,*d_dv;

__global__ void GPUInitOnDevice(double*u,double*v,int x,int y);
__global__ void GPUCalcBar(double*u,double*v,double*u0,double*v0,double*du,double*dv);
__global__ void GPUBoundary(double*u,double*v);
__global__ void GPUCalcSpace(double*u0,double*v0,double*du,double*dv);
__global__ void GPUPlaneWave(double*du,int ncount);
__global__ void GPUForEuler(double*u,double*u0,double*du,double*v,double*v0,double*dv);
__global__ void GPUUpdate(double*u,double*u0,double*v,double*v0);
__global__ void GPUTrancation(double*u,double*v);

void Manage_Memory(int phase,double**h_u,double**h_v,double**h_u0,double**h_v0,double**d_u,double**d_v,double**d_u0,double**d_v0,double**d_du,double**d_dv,double**h_du,double**h_dv){
	hipError_t Error;
	size_t size=(nx+1)*(ny+1)*sizeof(double);//index需要+1，因为使用了no flux 边界，需要往外扩大一条边。
	if (phase==1){
		*h_u=(double*)malloc(size);
		*h_v=(double*)malloc(size);
		*h_u0=(double*)malloc(size);
		*h_v0=(double*)malloc(size);
		*h_du=(double*)malloc(size);
		*h_dv=(double*)malloc(size);
		Error=hipMalloc((void**)d_u,size);
		Error=hipMalloc((void**)d_u0,size);
		Error=hipMalloc((void**)d_v,size);
		Error=hipMalloc((void**)d_v0,size);
		Error=hipMalloc((void**)d_du,size);
		Error=hipMalloc((void**)d_dv,size);
		printf("MemoryMalloc:%s\n",hipGetErrorString(Error));
	}
	if (phase==2){
		free(*h_u);
		free(*h_v);
		free(*h_u0);
		free(*h_v0);
		free(*h_du);
		free(*h_dv);
		Error=hipFree(*d_u);
		Error=hipFree(*d_u0);
		Error=hipFree(*d_v);
		Error=hipFree(*d_v0);
		Error=hipFree(*d_du);
		Error=hipFree(*d_dv);
	}

}

void Manage_Comms(int phase,double**h_u0,double**d_u0){

	hipError_t Error;
	size_t size=(nx+1)*(ny+1)*sizeof(double);
	if (phase==2){				
		Error=hipMemcpy(*h_u0,*d_u0,size,hipMemcpyDeviceToHost);
		printf("device to host:%s\n",hipGetErrorString(Error));
		}
	if (phase==1){
		Error=hipMemcpy(*d_u0,*h_u0,size,hipMemcpyHostToDevice);
		printf("host to device:%s\n",hipGetErrorString(Error));}
}
//----------------Init-------------------
__global__ void GPUInitOnDevice(double*u,double*v,int x,int y){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;//matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*x+ ix;//globalIdx
	//***init conditions*****			
		if(ix<(x+1)&&iy<(y+1)){
			u[idx] = 1.0; 
			v[idx] = 1.0;			
	}
}
void Call_GPU_Init(double**d_u0,double**d_v0,int x,int y){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUInitOnDevice<<<grid,block>>>(*d_u0,*d_v0,x,y);
	Error=hipDeviceSynchronize();
	printf("InitDeviceSynchronize:%s\n",hipGetErrorString(Error));
}
//----------------boundary-------------------
__global__ void GPUBoundary(double*u,double*v){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
		//****no flux boundary conditions*****
				u[iy*nx] = u[1+iy*nx];
				u[nx+iy*nx] = u[(nx-1)+iy*nx];
				v[iy*nx] = v[1+iy*nx];
				v[nx+iy*nx] = v[(nx-1)+iy*nx];
				u[ix] = u[ix+nx];
				u[ix+ny*nx] = u[ix+(ny-1)*nx];
				v[ix] = v[ix+nx];
				v[ix+ny*nx] = v[ix+(ny-1)*nx];		
}		
void Call_GPU_Boun(double**d_u0,double**d_v0){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUBoundary<<<grid,block>>>(*d_u0,*d_v0);
	Error=hipDeviceSynchronize();
	printf("GPUBoundarySynchronize:%s\n",hipGetErrorString(Error));
}			
			
//----------------Center Differnce for Space-------------------			
__global__ void GPUCalcSpace(double*u0,double*v0,double*du,double*dv){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx
		double sldx, sldy; 
		sldx = (1.0 / dx)/ dx;
		sldy = (1.0 / dy)/ dy;
		
		//if(ix>0&&iy>0){//此处需要控制边界
			du[idx] = -u0[idx]*(u0[idx] - 1.0)*(u0[idx] - (v0[idx] + bb) / aa) / eps + D*(sldx*(u0[(ix+1)+iy*nx] + u0[(ix-1)+iy*nx] - 2 * u0[ix+iy*nx]) 
									+ sldy*(u0[ix+(iy+1)*nx] + u0[ix+(iy-1)*nx] - 2 * u0[ix+iy*nx]));
			
			{
			if(u0[idx]< 1.0/3.0) dv[idx] = -v0[idx];
			else if(u0[idx]<1.0 || u0[idx]==1.0)
			dv[idx] = 1.0 - 6.75*u0[idx]*(u0[idx] - 1.0)*(u0[idx] - 1.0) - v0[idx];
			else dv[idx] = 1.0 - v0[idx];
						
			}	
		//}
}		
void Call_GPU_Space(double**d_u0,double**d_v0,double**d_du,double**d_dv){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUCalcSpace<<<grid,block>>>(*d_u0,*d_v0,*d_du,*d_dv);
	Error=hipDeviceSynchronize();
	printf("GPUCalcSpaceSynchronize:%s\n",hipGetErrorString(Error));
}			

//----------------stimulation with a plane waves-------------------		
__global__ void GPUPlaneWave(double*du,int ncount){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx	
		if (ncount < 10 && ix<3 ){
				du[idx] = du[idx] + 1.5*cos(w*ncount*h) + 0.5;
				
		}
}
void Call_GPU_PlaneWave(double**d_du,int ncount){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUPlaneWave<<<grid,block>>>(*d_du,ncount);
	Error=hipDeviceSynchronize();
	printf("GPUPlaneWaveSynchronize:%s\n",hipGetErrorString(Error));
}
//----------------Forward Euler for Time-------------------
__global__ void GPUForEuler(double*u,double*u0,double*du,double*v,double*v0,double*dv){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx
				printf("%18.7e",du[idx]);		
				u[idx] = u0[idx] + h*du[idx];
				v[idx] = v0[idx] + h*dv[idx];
}	
void Call_GPU_ForEuler(double**d_u,double**d_u0,double**d_du,double**d_v,double**d_v0,double**d_dv){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUForEuler<<<grid,block>>>(*d_u,*d_u0,*d_du,*d_v,*d_v0,*d_dv);
	Error=hipDeviceSynchronize();
	printf("GPUForEulerSynchronize:%s\n",hipGetErrorString(Error));
}			
//----------------Update the tow variabls-------------------
__global__ void GPUUpdate(double*u,double*u0,double*v,double*v0){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx	
				u0[idx] = u[idx];
				v0[idx] = v[idx];
}
void Call_GPU_Update(double**d_u,double**d_u0,double**d_v,double**d_v0){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUUpdate<<<grid,block>>>(*d_u,*d_u,*d_v,*d_v0);
	Error=hipDeviceSynchronize();
	printf("GPUUpdateSynchronize:%s\n",hipGetErrorString(Error));
}				
//----------------trancation 1/2 of the plane wave to generate a spiral wave-------------------
__global__ void GPUTrancation(double*u,double*v){
unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; //matrixIdx
unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;//matrixIdx
unsigned int idx = iy*nx+ ix;//globalIdx			
		if (idx<nx+1 && iy<(ny/2)+1){
				u[idx] = 0;
				v[idx] = 0;
		}
}
void Call_GPU_Trancation(double**d_u0,double**d_v0){	
	dim3 block(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid((nx+block.x-1)/block.x,(ny+block.y-1)/block.y);
	hipError_t Error;
	GPUTrancation<<<grid,block>>>(*d_u0,*d_v0);
	Error=hipDeviceSynchronize();
	printf("GPUTrancationSynchronize:%s\n",hipGetErrorString(Error));
}



